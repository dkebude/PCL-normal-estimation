#include "hip/hip_runtime.h"
#include <pcl/cuda/common/eigen.h>
#include <iostream>

// returns the current time
extern double get_time();

__global__ void compute(float *d_points_x, float *d_points_y, float *d_points_z, float *d_normal_x, float *d_normal_y, float *d_normal_z, int width, int height, int size)
{
  double centroid0 = 0;
  double centroid1 = 0;
  double centroid2 = 0;
  double C_00 = 0;
  double C_01 = 0;
  double C_02 = 0;
  double C_10 = 0;
  double C_11 = 0;
  double C_12 = 0;
  double C_20 = 0;
  double C_21 = 0;
  double C_22 = 0;

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = j*width+i;
  int NWidx = (j-1)*width+(i-1);
  int Nidx = (j-1)*width+i;
  int NEidx = (j-1)*width+(i+1);
  int Widx = j*width+(i-1);
  int Eidx = j*width+(i+1);
  int SWidx = (j+1)*width+(i-1);
  int Sidx = (j+1)*width+i;
  int SEidx = (j+1)*width+(i+1);

  if(i >= 0 && j >= 0 && j < height && i < width)
  {
    float NWx = d_points_x[NWidx];
    float NWy = d_points_y[NWidx];
    float NWz = d_points_z[NWidx];
    float Nx  = d_points_x[Nidx];
    float Ny  = d_points_y[Nidx];
    float Nz  = d_points_z[Nidx];
    float NEx = d_points_x[NEidx];
    float NEy = d_points_y[NEidx];
    float NEz = d_points_z[NEidx];
    float Wx  = d_points_x[Widx];
    float Wy  = d_points_y[Widx];
    float Wz  = d_points_z[Widx];
    float x   = d_points_x[idx];
    float y   = d_points_y[idx];
    float z   = d_points_z[idx];
    float Ex  = d_points_x[Eidx];
    float Ey  = d_points_y[Eidx];
    float Ez  = d_points_z[Eidx];
    float SWx = d_points_x[SWidx];
    float SWy = d_points_y[SWidx];
    float SWz = d_points_z[SWidx];
    float Sx  = d_points_x[Sidx];
    float Sy  = d_points_y[Sidx];
    float Sz  = d_points_z[Sidx];
    float SEx = d_points_x[SEidx];
    float SEy = d_points_y[SEidx];
    float SEz = d_points_z[SEidx];

    centroid0 = (NWx+Nx+NEx+Wx+x+Ex+SWx+Sx+SEx)/9;
    centroid1 = (NWy+Ny+NEy+Wy+y+Ey+SWy+Sy+SEy)/9;
    centroid2 = (NWz+Nz+NEz+Wz+z+Ez+SWz+Sz+SEz)/9;

    C_00 = ((NWx - centroid0)*(NWx - centroid0)
          + (Nx  - centroid0)*(Nx  - centroid0)
          + (NEx - centroid0)*(NEx - centroid0)
          + (Wx  - centroid0)*(Wx  - centroid0)
          + (x   - centroid0)*(x   - centroid0)
          + (Ex  - centroid0)*(Ex  - centroid0)
          + (SWx - centroid0)*(SWx - centroid0)
          + (Sx  - centroid0)*(Sx  - centroid0)
          + (SEx - centroid0)*(SEx - centroid0))/9;

    C_01 = ((NWx - centroid0)*(NWy - centroid1)
          + (Nx  - centroid0)*(Ny  - centroid1)
          + (NEx - centroid0)*(NEy - centroid1)
          + (Wx  - centroid0)*(Wy  - centroid1)
          + (x   - centroid0)*(y   - centroid1)
          + (Ex  - centroid0)*(Ey  - centroid1)
          + (SWx - centroid0)*(SWy - centroid1)
          + (Sx  - centroid0)*(Sy  - centroid1)
          + (SEx - centroid0)*(SEy - centroid1))/9;

    C_02 = ((NWx - centroid0)*(NWz - centroid2)
          + (Nx  - centroid0)*(Nz  - centroid2)
          + (NEx - centroid0)*(NEz - centroid2)
          + (Wx  - centroid0)*(Wz  - centroid2)
          + (x   - centroid0)*(z   - centroid2)
          + (Ex  - centroid0)*(Ez  - centroid2)
          + (SWx - centroid0)*(SWz - centroid2)
          + (Sx  - centroid0)*(Sz  - centroid2)
          + (SEx - centroid0)*(SEz - centroid2))/9;

    C_10 = C_01;
    
    C_11 = ((NWy - centroid1)*(NWy - centroid1)
          + (Ny  - centroid1)*(Ny  - centroid1)
          + (NEy - centroid1)*(NEy - centroid1)
          + (Wy  - centroid1)*(Wy  - centroid1)
          + (y   - centroid1)*(y   - centroid1)
          + (Ey  - centroid1)*(Ey  - centroid1)
          + (SWy - centroid1)*(SWy - centroid1)
          + (Sy  - centroid1)*(Sy  - centroid1)
          + (SEy - centroid1)*(SEy - centroid1))/9;
    
    C_12 = ((NWy - centroid1)*(NWz - centroid2)
         + (Ny   - centroid1)*(Nz  - centroid2)
         + (NEy  - centroid1)*(NEz - centroid2)
         + (Wy   - centroid1)*(Wz  - centroid2)
         + (y    - centroid1)*(z   - centroid2)
         + (Ey   - centroid1)*(Ez  - centroid2)
         + (SWy  - centroid1)*(SWz - centroid2)
         + (Sy   - centroid1)*(Sz  - centroid2)
         + (SEy  - centroid1)*(SEz - centroid2))/9;
    
    C_20 = C_02;
    
    C_21 = C_12;
    
    C_22 = ((NWz - centroid2)*(NWz - centroid2)
          + (Nz  - centroid2)*(Nz  - centroid2)
          + (NEz - centroid2)*(NEz - centroid2)
          + (Wz  - centroid2)*(Wz  - centroid2)
          + (z   - centroid2)*(z   - centroid2)
          + (Ez  - centroid2)*(Ez  - centroid2)
          + (SWz - centroid2)*(SWz - centroid2)
          + (Sz  - centroid2)*(Sz  - centroid2)
          + (SEz - centroid2)*(SEz - centroid2))/9;

    pcl::cuda::CovarianceMatrix C;
    C.data[0].x = C_00;
    C.data[0].y = C_01;
    C.data[0].z = C_02;
    C.data[1].x = C_10;
    C.data[1].y = C_11;
    C.data[1].z = C_12;
    C.data[2].x = C_20;
    C.data[2].y = C_21;
    C.data[2].z = C_22;
     
    float3 eigenvalue;
    pcl::cuda::CovarianceMatrix eigenvector;
    pcl::cuda::eigen33(C, eigenvector, eigenvalue);
    float3 vp;  
    vp.x = 1.17549e-38 - x;
    vp.y = 1.17549e-38 - y;
    vp.z = 1.17549e-38 - z;
    float3 normal = normalize(eigenvector.data[0]); 

    double flipDecision = dot(vp,normal);
    if(flipDecision < 0)
      normal *= -1;
    
    d_normal_x[idx] = normal.x;
    d_normal_y[idx] = normal.y;
    d_normal_z[idx] = normal.z;
  }
}

void kernel_wrapper(float *points_x, float *points_y, float *points_z, float *normal_x, float *normal_y, float *normal_z, int width, int height, int size, int block_size)
{
  double time_begin, time_end;

  float *d_points_x;
  float *d_points_y;
  float *d_points_z;
  float *d_normal_x;
  float *d_normal_y;
  float *d_normal_z;

  hipMalloc((void**) &d_points_x, (sizeof(float)*size));
  hipMalloc((void**) &d_points_y, (sizeof(float)*size));
  hipMalloc((void**) &d_points_z, (sizeof(float)*size));
  hipMalloc((void**) &d_normal_x, (sizeof(float)*size));
  hipMalloc((void**) &d_normal_y, (sizeof(float)*size));
  hipMalloc((void**) &d_normal_z, (sizeof(float)*size));
  hipMemcpy(d_points_x, points_x, (sizeof(float)*size), hipMemcpyHostToDevice);
  hipMemcpy(d_points_y, points_y, (sizeof(float)*size), hipMemcpyHostToDevice);
  hipMemcpy(d_points_z, points_z, (sizeof(float)*size), hipMemcpyHostToDevice);

  int ntx = block_size, nty = block_size;
  dim3 threads(ntx, nty);
  dim3 grid(width/ntx, height/nty);
  time_begin = get_time();
  compute<<<grid, threads>>>(d_points_x, d_points_y, d_points_z, d_normal_x, d_normal_y, d_normal_z, width, height, size);
  time_end = get_time();

  hipMemcpy(normal_x, d_normal_x, (sizeof(float)*size), hipMemcpyDeviceToHost);
  hipMemcpy(normal_y, d_normal_y, (sizeof(float)*size), hipMemcpyDeviceToHost);
  hipMemcpy(normal_z, d_normal_z, (sizeof(float)*size), hipMemcpyDeviceToHost);

  std::cout << "Time it took to compute: " << time_end - time_begin << std::endl;
}
